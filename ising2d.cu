#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <time.h>
#include "cutil.h"
#include <iostream>
#include <iomanip>
#include <hiprand.h>
#include <ctime>

#define FLAG_PRINT_SPINS 0
#define FLAG_ENERGY 0
#define T_START 2.30
#define T_FACTOR 0.001
#define T_END 2.20
#define GLOBAL_ITERATIONS 10
#define RANDOM_A 1664525
#define RANDOM_B 1013904223

#define BLOCK_SIZE 64

const unsigned int N=4*BLOCK_SIZE*BLOCK_SIZE;
const unsigned int n=2*BLOCK_SIZE;

/****
 *
 *  Function declaration
 *
 */
void calc(int argc,char** argv);
__global__ void device_function_main(int*,int*,int*,float,bool);

/****
 *
 *  Main function
 *
 */
int main(int argc,char** argv) {
  calc(argc,argv);
}

/****
 *
 *  Calc
 *
 */
void calc(int argc,char** argv) {

  std::cout << " -----------------------------------------------------------------------" << std::endl;
  std::cout <<" *" << std::endl;
  std::cout <<" *  GPU accelerated Monte Carlo simulation of the 2D Ising model" << std::endl;
  std::cout <<" *" <<  std::endl;
  std::cout <<" ----------------------------- Ising model ----------------------------- " << std::endl;
  std::cout <<" Number of Spins: " << N << std::endl;
  std::cout <<" Start Temperature: " << T_START <<std::endl;
  std::cout <<" Decreasing Factor: " << T_FACTOR <<std::endl;;
  std::cout <<" Final Temperature: " << T_END <<std::endl;
  std::cout <<" Global Iterations: " << GLOBAL_ITERATIONS << std::endl;

  //Init
  CUT_DEVICE_INIT(argc,argv);
  srand48(23);

  //Allocate and init host memory for output arrays
  int num_entries=0;
  for(double t=T_START; t>=T_END; t=t-T_FACTOR) num_entries++;
  unsigned int mem_out_size=sizeof(float)*num_entries;
  float* h_T=(float*) malloc(mem_out_size);
  float* h_E=(float*) malloc(mem_out_size);
  float* h_U=(float*) malloc(mem_out_size);
  unsigned int mem_ref_out_size=sizeof(double)*num_entries;
  double* h_ref_E=(double*) malloc(mem_ref_out_size);
  double* h_ref_U=(double*) malloc(mem_ref_out_size);
  num_entries=0;
  for(double t=T_START; t>=T_END; t=t-T_FACTOR) {
    h_T[num_entries]=t;
    num_entries++;
  }

  //Allocate and init host memory for simulation arrays
  unsigned int mem_size=sizeof(int)*N;
  unsigned int mem_size_random=sizeof(int)*BLOCK_SIZE*BLOCK_SIZE;
  int* h_random_data=(int*) malloc(mem_size_random);
  int* h_S=(int*) malloc(mem_size);
  unsigned int mem_size_out=sizeof(int)*BLOCK_SIZE;
  int* h_out=(int*) malloc(mem_size_out);
  h_random_data[0]=1;
  for(int i=1;i<BLOCK_SIZE*BLOCK_SIZE;i++) {
    h_random_data[i]=16807*h_random_data[i-1];
  }
  for(int i=0;i<N;i++) {
    if(drand48()>0.5) h_S[i]=-1;
    else h_S[i]=1;
  }

  //Create and start timer
  float gpu_sum=0;
  unsigned int timer=0;
  CUDA_SAFE_CALL(hipDeviceSynchronize());
  CUT_SAFE_CALL(cutCreateTimer(&timer));
  CUT_SAFE_CALL(cutStartTimer(timer));

  //Allocate device memory for arrays
  int* d_random_data;
  int* d_S;
  int* d_out;
  CUDA_SAFE_CALL(hipMalloc((void**) &d_random_data,mem_size_random));
  CUDA_SAFE_CALL(hipMalloc((void**) &d_S,mem_size));
  CUDA_SAFE_CALL(hipMalloc((void**) &d_out,mem_size_out));

  //Stop and destroy timer
  CUDA_SAFE_CALL(hipDeviceSynchronize());
  CUT_SAFE_CALL(cutStopTimer(timer));
  float gpu_dt_malloc=cutGetTimerValue(timer);
  gpu_sum+=gpu_dt_malloc;
  std::cout <<"\n --------------------------------- GPU --------------------------------- \n" <<std::endl;
  std::cout <<" Processing time on GPU for allocating:"<< std::setprecision(2)  << gpu_dt_malloc << "(ms)" <<std::endl;
  CUT_SAFE_CALL(cutDeleteTimer(timer));

  //Create and start timer
  timer=0;
  CUDA_SAFE_CALL(hipDeviceSynchronize());
  CUT_SAFE_CALL(cutCreateTimer(&timer));
  CUT_SAFE_CALL(cutStartTimer(timer));

  //Copy host memory to device and create mirror of d_S
  CUDA_SAFE_CALL(hipMemcpy(d_random_data,h_random_data,mem_size_random,hipMemcpyHostToDevice));
  CUDA_SAFE_CALL(hipMemcpy(d_S,h_S,mem_size,hipMemcpyHostToDevice));

  //Stop and destroy timer
  CUDA_SAFE_CALL(hipDeviceSynchronize());
  CUT_SAFE_CALL(cutStopTimer(timer));
  float gpu_dt_mem=cutGetTimerValue(timer);
  gpu_sum+=gpu_dt_mem;
  printf(" Processing time on GPU for memory transfer: %f (ms) \n",gpu_dt_mem);
  CUT_SAFE_CALL(cutDeleteTimer(timer));

  //Print spins
  if(FLAG_PRINT_SPINS) {
    CUDA_SAFE_CALL(hipMemcpy(h_S,d_S,mem_size,hipMemcpyDeviceToHost));
    for(int i=0;i<BLOCK_SIZE*2;i++) {
      for(int j=0;j<BLOCK_SIZE*2;j++) {
	if(h_S[i*BLOCK_SIZE*2+j]>0) printf("+ ");
	else printf("- ");
      }
      printf("\n");
    }
    printf("\n");
  }

  //Create and start timer
  timer=0;
  CUDA_SAFE_CALL(hipDeviceSynchronize());
  CUT_SAFE_CALL(cutCreateTimer(&timer));
  CUT_SAFE_CALL(cutStartTimer(timer));

  //Calc energy
  num_entries=0;
  dim3 threads(BLOCK_SIZE);
  dim3 grid(BLOCK_SIZE);

  FILE *fp = fopen("gpu_T_U.dat","w");
  for(float t=T_START;t>=T_END;t=t-T_FACTOR) {
    double avg_H=0;
    double avg_H_2=0;
    double avg_H_4=0;
    for(int global_iteration=0;global_iteration<GLOBAL_ITERATIONS;global_iteration++) {
      device_function_main<<<grid,threads>>>(d_S,d_out,d_random_data,t,true);
      device_function_main<<<grid,threads>>>(d_S,d_out,d_random_data,t,false);

      CUDA_SAFE_CALL(hipMemcpy(h_out,d_out,mem_size_out,hipMemcpyDeviceToHost));
      int energy_sum=0;
      for(int i=0;i<BLOCK_SIZE;i++) energy_sum+=h_out[i];
      avg_H+=(float)energy_sum/N;
      avg_H_2+=pow((float)energy_sum/N,2);
      avg_H_4+=pow((float)energy_sum/N,4);
     // printf("%f\n",(float)energy_sum/N);
    }
    h_E[num_entries]=avg_H/GLOBAL_ITERATIONS;
    h_U[num_entries]=1.0-((avg_H_4/GLOBAL_ITERATIONS)/(3*pow(avg_H_2/GLOBAL_ITERATIONS,2)));
    //h_U[num_entries]=0.5*(3-(avg_H_4/GLOBAL_ITERATIONS)/(3*pow(avg_H_2/GLOBAL_ITERATIONS,2)));
    //printf("%f %f %f\n",h_T[num_entries],h_E[num_entries],h_U[num_entries]);
    std::cout << h_T[num_entries] << " " << h_E[num_entries] << " " << h_U[num_entries] << std::endl;
    fprintf(fp,"%f %f\n",h_T[num_entries],h_U[num_entries]);
    printf("\n");
    num_entries++;
  }
  fclose(fp);


  //Stop and destroy timer
  CUDA_SAFE_CALL(hipDeviceSynchronize());
  CUT_SAFE_CALL(cutStopTimer(timer));
  float gpu_dt_main=cutGetTimerValue(timer);
  gpu_sum+=gpu_dt_main;
  std::cout << " Processing time on GPU for main function: "<< std::setprecision(2) <<std::fixed << gpu_dt_main << "(ms)" << std::endl;
  std::cout <<" Total processing time on GPU:"<< std::setprecision(2) <<std::fixed  << gpu_sum << "(ms)" << std::endl;
  CUT_SAFE_CALL(cutDeleteTimer(timer));

  //Check kernel execution
  CUT_CHECK_ERROR("Kernel execution failed");

  //Print spins
  if(FLAG_PRINT_SPINS) {
    CUDA_SAFE_CALL(hipMemcpy(h_S,d_S,mem_size,hipMemcpyDeviceToHost));
    for(int i=0;i<BLOCK_SIZE*2;i++) {
      for(int j=0;j<BLOCK_SIZE*2;j++) {
	if(h_S[i*BLOCK_SIZE*2+j]>0) printf("+ ");
	else printf("- ");
      }
      printf("\n");
    }
  }

  //Create and start timer
  timer=0;
  CUDA_SAFE_CALL(hipDeviceSynchronize());
  CUT_SAFE_CALL(cutCreateTimer(&timer));
  CUT_SAFE_CALL(cutStartTimer(timer));

  //Reference solution
  //cpu_function(h_ref_E, h_ref_U, h_S);

  //Print spins
  if(FLAG_PRINT_SPINS) {
    printf("\n");
    for(int i=0;i<BLOCK_SIZE*2;i++) {
      for(int j=0;j<BLOCK_SIZE*2;j++) {
	if(h_S[i*BLOCK_SIZE*2+j]>0) printf("+1 ");
	else printf("-1 ");
      }
      printf("\n");
    }
  }

  //Stop and destroy timer
  CUDA_SAFE_CALL(hipDeviceSynchronize());
  CUT_SAFE_CALL(cutStopTimer(timer));
  float cpu_sum=cutGetTimerValue(timer);
  printf("\n --------------------------------- CPU --------------------------------- \n");
  printf(" Total processing time on CPU: %f (ms) \n",cpu_sum);
  CUT_SAFE_CALL(cutDeleteTimer(timer));
  printf("\n Speedup: %fX \n\n",(cpu_sum/gpu_sum));

  //Cleaning memory
  free(h_T);
  free(h_U);
  free(h_E);
  free(h_ref_E);
  free(h_ref_U);
  free(h_random_data);
  free(h_S);
  free(h_out);
  CUDA_SAFE_CALL(hipFree(d_random_data));
  CUDA_SAFE_CALL(hipFree(d_S));
  CUDA_SAFE_CALL(hipFree(d_out));
}

/****
 *
 *  Device function main
 *
 */
__global__ void device_function_main(int* S,int* out,int* R,float t,bool flag) {

  //Energy variable
  int dH=0;
  float exp_dH_4=exp(-(4.0)/t);
  float exp_dH_8=exp(-(8.0)/t);

  //Allocate shared memory
  __shared__ int r[BLOCK_SIZE];

  //Load random data
  r[threadIdx.x]=R[threadIdx.x+BLOCK_SIZE*blockIdx.x];
  __syncthreads();

  if(flag) {

    //Create new random numbers
    r[threadIdx.x]=RANDOM_A*r[threadIdx.x]+RANDOM_B;

    //Spin update top left
    if(blockIdx.x==0) { //Top
      if(threadIdx.x==0) { //Left
        dH=2*S[2*threadIdx.x]*(
                               S[2*threadIdx.x+1]+
                               S[2*threadIdx.x-1+2*BLOCK_SIZE]+
                               S[2*threadIdx.x+2*BLOCK_SIZE]+
                               S[2*threadIdx.x+N-2*BLOCK_SIZE]);
      }
      else {
        dH=2*S[2*threadIdx.x]*(
                               S[2*threadIdx.x+1]+
                               S[2*threadIdx.x-1]+
                               S[2*threadIdx.x+2*BLOCK_SIZE]+
                               S[2*threadIdx.x+N-2*BLOCK_SIZE]);
      }
    }
    else {
      if(threadIdx.x==0) { //Left
        dH=2*S[2*threadIdx.x+4*BLOCK_SIZE*blockIdx.x]*(
                                                       S[2*threadIdx.x+4*BLOCK_SIZE*blockIdx.x+1]+
                                                       S[2*threadIdx.x+4*BLOCK_SIZE*blockIdx.x-1+2*BLOCK_SIZE]+
                                                       S[2*threadIdx.x+4*BLOCK_SIZE*blockIdx.x+2*BLOCK_SIZE]+
                                                       S[2*threadIdx.x+4*BLOCK_SIZE*blockIdx.x-2*BLOCK_SIZE]);
      }
      else {
        dH=2*S[2*threadIdx.x+4*BLOCK_SIZE*blockIdx.x]*(
                                                       S[2*threadIdx.x+4*BLOCK_SIZE*blockIdx.x+1]+
                                                       S[2*threadIdx.x+4*BLOCK_SIZE*blockIdx.x-1]+
                                                       S[2*threadIdx.x+4*BLOCK_SIZE*blockIdx.x+2*BLOCK_SIZE]+
                                                       S[2*threadIdx.x+4*BLOCK_SIZE*blockIdx.x-2*BLOCK_SIZE]);
      }
    }

    if(dH==4) {
      if(fabs(r[threadIdx.x]*4.656612e-10)<exp_dH_4) {
        S[2*threadIdx.x+4*BLOCK_SIZE*blockIdx.x]=-S[2*threadIdx.x+4*BLOCK_SIZE*blockIdx.x];
      }
    }
    else if(dH==8) {
      if(fabs(r[threadIdx.x]*4.656612e-10)<exp_dH_8) {
        S[2*threadIdx.x+4*BLOCK_SIZE*blockIdx.x]=-S[2*threadIdx.x+4*BLOCK_SIZE*blockIdx.x];
      }
    }
    else {
      S[2*threadIdx.x+4*BLOCK_SIZE*blockIdx.x]=-S[2*threadIdx.x+4*BLOCK_SIZE*blockIdx.x];
    }

    //Create new random numbers
    r[threadIdx.x]=RANDOM_A*r[threadIdx.x]+RANDOM_B;

    //Spin update bottom right
    if(blockIdx.x==BLOCK_SIZE-1) { //Bottom
      if(threadIdx.x==BLOCK_SIZE-1) { //Right
        dH=2*S[2*threadIdx.x+1+4*BLOCK_SIZE*blockIdx.x+2*BLOCK_SIZE]*(
                                                                      S[2*threadIdx.x+4*BLOCK_SIZE*blockIdx.x+2]+
                                                                      S[2*threadIdx.x+4*BLOCK_SIZE*blockIdx.x+2*BLOCK_SIZE]+
                                                                      S[2*threadIdx.x+1]+
                                                                      S[2*threadIdx.x+1+4*BLOCK_SIZE*blockIdx.x]);
      }
      else {
        dH=2*S[2*threadIdx.x+1+4*BLOCK_SIZE*blockIdx.x+2*BLOCK_SIZE]*(
                                                                      S[2*threadIdx.x+4*BLOCK_SIZE*blockIdx.x+2*BLOCK_SIZE+2]+
                                                                      S[2*threadIdx.x+4*BLOCK_SIZE*blockIdx.x+2*BLOCK_SIZE]+
                                                                      S[2*threadIdx.x+1]+
                                                                      S[2*threadIdx.x+1+4*BLOCK_SIZE*blockIdx.x]);
      }
    }
    else {
      if(threadIdx.x==BLOCK_SIZE-1) { //Right
        dH=2*S[2*threadIdx.x+1+4*BLOCK_SIZE*blockIdx.x+2*BLOCK_SIZE]*(
                                                                      S[2*threadIdx.x+4*BLOCK_SIZE*blockIdx.x+2]+
                                                                      S[2*threadIdx.x+4*BLOCK_SIZE*blockIdx.x+2*BLOCK_SIZE]+
                                                                      S[2*threadIdx.x+1+4*BLOCK_SIZE*(blockIdx.x+1)]+
                                                                      S[2*threadIdx.x+1+4*BLOCK_SIZE*blockIdx.x]);
      }
      else {
        dH=2*S[2*threadIdx.x+1+4*BLOCK_SIZE*blockIdx.x+2*BLOCK_SIZE]*(
                                                                      S[2*threadIdx.x+4*BLOCK_SIZE*blockIdx.x+2*BLOCK_SIZE+2]+
                                                                      S[2*threadIdx.x+4*BLOCK_SIZE*blockIdx.x+2*BLOCK_SIZE]+
                                                                      S[2*threadIdx.x+1+4*BLOCK_SIZE*(blockIdx.x+1)]+
                                                                      S[2*threadIdx.x+1+4*BLOCK_SIZE*blockIdx.x]);
      }
    }

    if(dH==4) {
      if(fabs(r[threadIdx.x]*4.656612e-10)<exp_dH_4) {
        S[2*threadIdx.x+1+4*BLOCK_SIZE*blockIdx.x+2*BLOCK_SIZE]=-S[2*threadIdx.x+1+4*BLOCK_SIZE*blockIdx.x+2*BLOCK_SIZE];
      }
    }
    else if(dH==8) {
      if(fabs(r[threadIdx.x]*4.656612e-10)<exp_dH_8) {
        S[2*threadIdx.x+1+4*BLOCK_SIZE*blockIdx.x+2*BLOCK_SIZE]=-S[2*threadIdx.x+1+4*BLOCK_SIZE*blockIdx.x+2*BLOCK_SIZE];
      }
    }
    else {
      S[2*threadIdx.x+1+4*BLOCK_SIZE*blockIdx.x+2*BLOCK_SIZE]=-S[2*threadIdx.x+1+4*BLOCK_SIZE*blockIdx.x+2*BLOCK_SIZE];
    }

    __syncthreads();

  }
  else {

    //Create new random numbers
    r[threadIdx.x]=RANDOM_A*r[threadIdx.x]+RANDOM_B;

    //Spin update top right
    if(blockIdx.x==0) { //Top
      if(threadIdx.x==BLOCK_SIZE-1) { //Right
        dH=2*S[2*threadIdx.x+1]*(
                                 S[2*threadIdx.x+2-2*BLOCK_SIZE]+
                                 S[2*threadIdx.x]+
                                 S[2*threadIdx.x+1+2*BLOCK_SIZE]+
                                 S[2*threadIdx.x+1+N-2*BLOCK_SIZE]);
      }
      else {
        dH=2*S[2*threadIdx.x+1]*(
                                 S[2*threadIdx.x+2]+
                                 S[2*threadIdx.x]+
                                 S[2*threadIdx.x+1+2*BLOCK_SIZE]+
                                 S[2*threadIdx.x+1+N-2*BLOCK_SIZE]);
      }
    }
    else {
      if(threadIdx.x==BLOCK_SIZE-1) { //Right
        dH=2*S[2*threadIdx.x+1+4*BLOCK_SIZE*blockIdx.x]*(
                                                         S[2*threadIdx.x+4*BLOCK_SIZE*blockIdx.x+2-2*BLOCK_SIZE]+
                                                         S[2*threadIdx.x+4*BLOCK_SIZE*blockIdx.x]+
                                                         S[2*threadIdx.x+1+4*BLOCK_SIZE*blockIdx.x+2*BLOCK_SIZE]+
                                                         S[2*threadIdx.x+1+4*BLOCK_SIZE*blockIdx.x-2*BLOCK_SIZE]);
      }
      else {
        dH=2*S[2*threadIdx.x+1+4*BLOCK_SIZE*blockIdx.x]*(
                                                         S[2*threadIdx.x+4*BLOCK_SIZE*blockIdx.x+2]+
                                                         S[2*threadIdx.x+4*BLOCK_SIZE*blockIdx.x]+
                                                         S[2*threadIdx.x+1+4*BLOCK_SIZE*blockIdx.x+2*BLOCK_SIZE]+
                                                         S[2*threadIdx.x+1+4*BLOCK_SIZE*blockIdx.x-2*BLOCK_SIZE]);
      }
    }

    if(dH==4) {
      if(fabs(r[threadIdx.x]*4.656612e-10)<exp_dH_4) {
        S[2*threadIdx.x+1+4*BLOCK_SIZE*blockIdx.x]=-S[2*threadIdx.x+1+4*BLOCK_SIZE*blockIdx.x];
      }
    }
    else if(dH==8) {
      if(fabs(r[threadIdx.x]*4.656612e-10)<exp_dH_8) {
        S[2*threadIdx.x+1+4*BLOCK_SIZE*blockIdx.x]=-S[2*threadIdx.x+1+4*BLOCK_SIZE*blockIdx.x];
      }
    }
    else {
      S[2*threadIdx.x+1+4*BLOCK_SIZE*blockIdx.x]=-S[2*threadIdx.x+1+4*BLOCK_SIZE*blockIdx.x];
    }

    //Create new random numbers
    r[threadIdx.x]=RANDOM_A*r[threadIdx.x]+RANDOM_B;

    //Spin update bottom left
    if(blockIdx.x==BLOCK_SIZE-1) { //Bottom
      if(threadIdx.x==0) { //Left
        dH=2*S[2*threadIdx.x+4*BLOCK_SIZE*blockIdx.x+2*BLOCK_SIZE]*(
                                                                    S[2*threadIdx.x+4*BLOCK_SIZE*blockIdx.x+2*BLOCK_SIZE+1]+
                                                                    S[2*threadIdx.x+4*BLOCK_SIZE*(blockIdx.x+1)-1]+
                                                                    S[2*threadIdx.x]+
                                                                    S[2*threadIdx.x+4*BLOCK_SIZE*blockIdx.x]);
      }
      else {
        dH=2*S[2*threadIdx.x+4*BLOCK_SIZE*blockIdx.x+2*BLOCK_SIZE]*(
                                                                    S[2*threadIdx.x+4*BLOCK_SIZE*blockIdx.x+2*BLOCK_SIZE+1]+
                                                                    S[2*threadIdx.x+4*BLOCK_SIZE*blockIdx.x+2*BLOCK_SIZE-1]+
                                                                    S[2*threadIdx.x]+
                                                                    S[2*threadIdx.x+4*BLOCK_SIZE*blockIdx.x]);
      }
    }
    else {
      if(threadIdx.x==0) { //Left
        dH=2*S[2*threadIdx.x+4*BLOCK_SIZE*blockIdx.x+2*BLOCK_SIZE]*(
                                                                    S[2*threadIdx.x+4*BLOCK_SIZE*blockIdx.x+2*BLOCK_SIZE+1]+
                                                                    S[2*threadIdx.x+4*BLOCK_SIZE*(blockIdx.x+1)-1]+
                                                                    S[2*threadIdx.x+4*BLOCK_SIZE*(blockIdx.x+1)]+
                                                                    S[2*threadIdx.x+4*BLOCK_SIZE*blockIdx.x]);
      }
      else {
        dH=2*S[2*threadIdx.x+4*BLOCK_SIZE*blockIdx.x+2*BLOCK_SIZE]*(
                                                                    S[2*threadIdx.x+4*BLOCK_SIZE*blockIdx.x+2*BLOCK_SIZE+1]+
                                                                    S[2*threadIdx.x+4*BLOCK_SIZE*blockIdx.x+2*BLOCK_SIZE-1]+
                                                                    S[2*threadIdx.x+4*BLOCK_SIZE*(blockIdx.x+1)]+
                                                                    S[2*threadIdx.x+4*BLOCK_SIZE*blockIdx.x]);
      }
    }

    if(dH==4) {
      if(fabs(r[threadIdx.x]*4.656612e-10)<exp_dH_4) {
        S[2*threadIdx.x+4*BLOCK_SIZE*blockIdx.x+2*BLOCK_SIZE]=-S[2*threadIdx.x+4*BLOCK_SIZE*blockIdx.x+2*BLOCK_SIZE];
      }
    }
    else if(dH==8) {
      if(fabs(r[threadIdx.x]*4.656612e-10)<exp_dH_8) {
        S[2*threadIdx.x+4*BLOCK_SIZE*blockIdx.x+2*BLOCK_SIZE]=-S[2*threadIdx.x+4*BLOCK_SIZE*blockIdx.x+2*BLOCK_SIZE];
      }
    }
    else {
      S[2*threadIdx.x+4*BLOCK_SIZE*blockIdx.x+2*BLOCK_SIZE]=-S[2*threadIdx.x+4*BLOCK_SIZE*blockIdx.x+2*BLOCK_SIZE];
    }

  }

  //Transfer random data back to global memory
  R[threadIdx.x+BLOCK_SIZE*blockIdx.x]=r[threadIdx.x];

  if(!flag) {

    //For reduction shared memory array r is used
    if(FLAG_ENERGY) {

      //Calc energy
      if(blockIdx.x==BLOCK_SIZE-1) { //Bottom
	if(threadIdx.x==BLOCK_SIZE-1) { //Right
	  dH=-S[2*threadIdx.x+4*BLOCK_SIZE*blockIdx.x]*(S[2*threadIdx.x+4*BLOCK_SIZE*blockIdx.x+1]+S[2*threadIdx.x+4*BLOCK_SIZE*blockIdx.x+2*BLOCK_SIZE])
	    -S[2*threadIdx.x+1+4*BLOCK_SIZE*blockIdx.x]*(S[2*threadIdx.x+1+4*BLOCK_SIZE*blockIdx.x+1-2*BLOCK_SIZE]+S[2*threadIdx.x+1+4*BLOCK_SIZE*blockIdx.x+2*BLOCK_SIZE])
	    -S[2*threadIdx.x+4*BLOCK_SIZE*blockIdx.x+2*BLOCK_SIZE]*(S[2*threadIdx.x+4*BLOCK_SIZE*blockIdx.x+1+2*BLOCK_SIZE]+S[2*threadIdx.x])
	    -S[2*threadIdx.x+1+4*BLOCK_SIZE*blockIdx.x+2*BLOCK_SIZE]*(S[2*threadIdx.x+4*BLOCK_SIZE*blockIdx.x+2]+S[2*threadIdx.x+1]);
	}
	else {
	  dH=-S[2*threadIdx.x+4*BLOCK_SIZE*blockIdx.x]*(S[2*threadIdx.x+4*BLOCK_SIZE*blockIdx.x+1]+S[2*threadIdx.x+4*BLOCK_SIZE*blockIdx.x+2*BLOCK_SIZE])
	    -S[2*threadIdx.x+1+4*BLOCK_SIZE*blockIdx.x]*(S[2*threadIdx.x+1+4*BLOCK_SIZE*blockIdx.x+1]+S[2*threadIdx.x+1+4*BLOCK_SIZE*blockIdx.x+2*BLOCK_SIZE])
	    -S[2*threadIdx.x+4*BLOCK_SIZE*blockIdx.x+2*BLOCK_SIZE]*(S[2*threadIdx.x+4*BLOCK_SIZE*blockIdx.x+1+2*BLOCK_SIZE]+S[2*threadIdx.x])
	    -S[2*threadIdx.x+1+4*BLOCK_SIZE*blockIdx.x+2*BLOCK_SIZE]*(S[2*threadIdx.x+4*BLOCK_SIZE*blockIdx.x+2+2*BLOCK_SIZE]+S[2*threadIdx.x+1]);
	}
      }
      else {
	if(threadIdx.x==BLOCK_SIZE-1) { //Right
	  dH=-S[2*threadIdx.x+4*BLOCK_SIZE*blockIdx.x]*(S[2*threadIdx.x+4*BLOCK_SIZE*blockIdx.x+1]+S[2*threadIdx.x+4*BLOCK_SIZE*blockIdx.x+2*BLOCK_SIZE])
	    -S[2*threadIdx.x+1+4*BLOCK_SIZE*blockIdx.x]*(S[2*threadIdx.x+1+4*BLOCK_SIZE*blockIdx.x+1-2*BLOCK_SIZE]+S[2*threadIdx.x+1+4*BLOCK_SIZE*blockIdx.x+2*BLOCK_SIZE])
	    -S[2*threadIdx.x+4*BLOCK_SIZE*blockIdx.x+2*BLOCK_SIZE]*(S[2*threadIdx.x+4*BLOCK_SIZE*blockIdx.x+1+2*BLOCK_SIZE]+S[2*threadIdx.x+4*BLOCK_SIZE*(blockIdx.x+1)])
	    -S[2*threadIdx.x+1+4*BLOCK_SIZE*blockIdx.x+2*BLOCK_SIZE]*(S[2*threadIdx.x+4*BLOCK_SIZE*blockIdx.x+2]+S[2*threadIdx.x+1+4*BLOCK_SIZE*(blockIdx.x+1)]);
	}
	else {
	  dH=-S[2*threadIdx.x+4*BLOCK_SIZE*blockIdx.x]*(S[2*threadIdx.x+4*BLOCK_SIZE*blockIdx.x+1]+S[2*threadIdx.x+4*BLOCK_SIZE*blockIdx.x+2*BLOCK_SIZE])
	    -S[2*threadIdx.x+1+4*BLOCK_SIZE*blockIdx.x]*(S[2*threadIdx.x+1+4*BLOCK_SIZE*blockIdx.x+1]+S[2*threadIdx.x+1+4*BLOCK_SIZE*blockIdx.x+2*BLOCK_SIZE])
	    -S[2*threadIdx.x+4*BLOCK_SIZE*blockIdx.x+2*BLOCK_SIZE]*(S[2*threadIdx.x+4*BLOCK_SIZE*blockIdx.x+1+2*BLOCK_SIZE]+S[2*threadIdx.x+4*BLOCK_SIZE*(blockIdx.x+1)])
	    -S[2*threadIdx.x+1+4*BLOCK_SIZE*blockIdx.x+2*BLOCK_SIZE]*(S[2*threadIdx.x+4*BLOCK_SIZE*blockIdx.x+2+2*BLOCK_SIZE]+S[2*threadIdx.x+1+4*BLOCK_SIZE*(blockIdx.x+1)]);
	}
      }
      __syncthreads();

    }
    else {

      //Calc magnetisation
      dH=S[2*threadIdx.x+4*BLOCK_SIZE*blockIdx.x]
	+S[2*threadIdx.x+1+4*BLOCK_SIZE*blockIdx.x]
	+S[2*threadIdx.x+4*BLOCK_SIZE*blockIdx.x+2*BLOCK_SIZE]
	+S[2*threadIdx.x+1+4*BLOCK_SIZE*blockIdx.x+2*BLOCK_SIZE];
      __syncthreads();
    }

    //Save partial results back to shared memory in new structure
    r[threadIdx.x]=dH;

    //Reduction on GPU
    for(unsigned int dx=1;dx<BLOCK_SIZE;dx*=2) {
      if(threadIdx.x%(2*dx)==0) {
	r[threadIdx.x]+=r[threadIdx.x+dx];
      }
      __syncthreads();
    }

    //Save in out
    if(threadIdx.x==0) out[blockIdx.x]=r[0];
  }
}


